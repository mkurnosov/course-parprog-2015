#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <inttypes.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

const int block_size = 1024;
const int n = 4 * (1 << 20);

void reduce_cpu(int *v, int n, int *sum)
{
    /*
    int s = 0.0;
    for (int i = 0; i < n; i++)
        s += v[i];        
    *sum = s;    
    */

    // Kahan's summation algorithm
    int s = v[0];
    int c = (int)0.0;

    for (int i = 1; i < n; i++) {
        int y = v[i] - c;
        int t = s + y;
        c = (t - s) - y;
        s = t;
    }
    *sum = s;
}

__global__ void reduce_per_block(int *v, int n, int *per_block_sum)
{
    __shared__ int sdata[block_size];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
        
    if (i < n) {
        sdata[tid] = v[i];
        __syncthreads();
        
        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s)
                sdata[tid] += sdata[tid + s];
            __syncthreads();
        }
        if (tid == 0)
            per_block_sum[blockIdx.x] = sdata[0];
    }
}

double wtime()
{
    struct timeval t;
    gettimeofday(&t, NULL);
    return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}

int main()
{
    double tcpu = 0, tgpu = 0, tmem = 0;
    
    size_t size = sizeof(int) * n;
    int *v = (int *)malloc(size);

    srand(0);
    for (size_t i = 0; i < n; i++)
        v[i] = i + 1;

    int sum;
    tcpu = -wtime();
    reduce_cpu(v, n, &sum);
    tcpu += wtime();
    
    /* Allocate on device */
    int threads_per_block = block_size;
    int blocks = (n + threads_per_block - 1) / threads_per_block;

    int *dv;    
    int *per_block_sum;
    int *sums = (int *)malloc(sizeof(int) * blocks);
    tmem = -wtime();
    hipMalloc((void **)&per_block_sum, sizeof(int) * blocks);
    hipMalloc((void **)&dv, size);
    hipMemcpy(dv, v, size, hipMemcpyHostToDevice);
    tmem += wtime();
        
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocks, threads_per_block);
    fflush(stdout);
    
    /* Compute per block sum */
    tgpu = -wtime();
    reduce_per_block<<<blocks, threads_per_block>>>(dv, n, per_block_sum);
    hipDeviceSynchronize();
    tgpu += wtime();    

    tmem = -wtime();
    hipMemcpy(sums, per_block_sum, sizeof(int) * blocks, hipMemcpyDeviceToHost);
    tmem += wtime();

    /* Compute block sum */
    tgpu -= wtime();
    int sum_gpu = 0;
    for (int i = 0; i < blocks; i++)
        sum_gpu += sums[i];
    tgpu += wtime();
    
    printf("Sum (CPU) = %d\n", sum);
    printf("Sum (GPU) = %d\n", sum_gpu);

    printf("CPU version (sec.): %.6f\n", tcpu);
    printf("GPU version (sec.): %.6f\n", tgpu);
    printf("GPU bandwidth (GiB/s): %.2f\n", 1.0e-9 * size / (tgpu + tmem));
    printf("Speedup: %.2f\n", tcpu / tgpu);
    printf("Speedup (with mem ops.): %.2f\n", tcpu / (tgpu + tmem));

    hipFree(per_block_sum);
    hipFree(dv);
    free(sums);
    free(v);
    hipDeviceReset();
    return 0;
}

