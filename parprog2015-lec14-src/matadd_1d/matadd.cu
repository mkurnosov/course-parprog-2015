/*
 * matadd.cu:
 *
 */

#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

enum {
    ROWS = 1024,
    COLS = 1024
};

__global__ void matadd(const float *a, const float *b, float *c, int m, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < m * n)
        c[idx] = a[idx] + b[idx];
}

void matadd_host(float *a, float *b, float *c, int m, int n)
{
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            int idx = i * n + j;
            c[idx] = a[idx] + b[idx];
        }            
    }
}

double wtime()
{
    struct timeval t;
    gettimeofday(&t, NULL);
    return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}

int main()
{
    double tcpu, tgpu, tmem;
    hipError_t err;
    
    /* Allocate memory on host */
    size_t size = sizeof(float) * ROWS * COLS;
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Allocation error.\n");
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < ROWS * COLS; i++) {
        h_A[i] = 1.5;
        h_B[i] = 2.5;
    }
    
    tcpu = -wtime();
    matadd_host(h_A, h_B, h_C, ROWS, COLS);
    tcpu += wtime();

    // Verify that the result vector is correct
    for (int i = 0; i < ROWS * COLS; i++) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "CPU results verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    
    /* Allocate vectors on device */
    float *d_A = NULL, *d_B = NULL,  *d_C = NULL;
    if (hipMalloc((void **)&d_A, size) != hipSuccess) {
        fprintf(stderr, "Allocation error\n");
        exit(EXIT_FAILURE);
    }
    if (hipMalloc((void **)&d_B, size) != hipSuccess) {
        fprintf(stderr, "Allocation error\n");
        exit(EXIT_FAILURE);
    }
    if (hipMalloc((void **)&d_C, size) != hipSuccess) {
        fprintf(stderr, "Allocation error\n");
        exit(EXIT_FAILURE);
    }

    /* Copy the host vectors to device */
    tmem = -wtime();
    if (hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Host to device copying failed\n");
        exit(EXIT_FAILURE);        
    }
    if (hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Host to device copying failed\n");
        exit(EXIT_FAILURE);        
    }
    tmem += wtime();
    
    /* Launch the kernel */
    int threadsPerBlock = 1024;
    int blocksPerGrid =(ROWS * COLS + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    tgpu = -wtime();
    matadd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, ROWS, COLS);
    hipDeviceSynchronize();
    tgpu += wtime();
    
    if ( (err = hipGetLastError()) != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    /* Copy the device vectors to host */
    tmem -= wtime();
    if (hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "Device to host copying failed\n");
        exit(EXIT_FAILURE);
    }
    tmem += wtime();

    // Verify that the result vector is correct
    for (int i = 0; i < ROWS * COLS; i++) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "GPU results verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    
    printf("CPU version (sec.): %.6f\n", tcpu);
    printf("GPU version (sec.): %.6f\n", tgpu);
    printf("Memory ops. (sec.): %.6f\n", tmem);
    printf("Memory bw. (MiB/sec.): %.2f\n", ((3 * size) >> 20) / tmem);
    printf("CPU perf (MFLOPS): %.2f\n", (ROWS * COLS >> 20) / tcpu);
    printf("GPU perf (MFLOPS): %.2f\n", (ROWS * COLS >> 20) / tgpu);
    printf("Speedup: %.2f\n", tcpu / tgpu);
    printf("Speedup (with mem ops.): %.2f\n", tcpu / (tgpu + tmem));

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    hipDeviceReset();
    return 0;
}

