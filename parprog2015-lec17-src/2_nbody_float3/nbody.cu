#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

const float eps = 0.0001f;
const float dt = 0.01f;
const int N = 128 * 1024;

#define coord float3

__global__ void integrate(coord *new_p, coord *new_v, coord *p, coord *v, 
                          int n, float dt)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;    
    if (index >= n)
        return;
    
    coord body_pos = p[index];      
    coord body_vel = v[index];      
    coord f;
    f.x = 0;
    f.y = 0;
    f.z = 0;

    for (int i = 0; i < n; i++) {
        coord pi = p[i];            
        coord r;
        // Vector from p[i] to body
        r.x = pi.x - body_pos.x;    
        r.y = pi.y - body_pos.y;    
        r.z = pi.z - body_pos.z;    
        
        float invDist = 1.0 / sqrtf(r.x * r.x + r.y * r.y + r.z * r.z + eps * eps);
        float s = invDist * invDist * invDist;
        // Add force of body i
        f.x += r.x * s;
        f.y += r.y * s;
        f.z += r.z * s;
    }
    
    // Correct velocity
    body_vel.x += f.x * dt;
    body_vel.y += f.y * dt;
    body_vel.z += f.z * dt;
    body_pos.x += body_vel.x * dt;
    body_pos.y += body_vel.y * dt;
    body_pos.z += body_vel.z * dt;
    
    new_p[index] = body_pos;
    new_v[index] = body_vel;
}

double wtime()
{
    struct timeval t;
    gettimeofday(&t, NULL);
    return (double)t.tv_sec + (double)t.tv_usec * 1E-6;
}

void init_rand(coord *v, int n)
{
    for (int i = 0; i < n; i++) {
        v[i].x = rand() / (float)RAND_MAX - 0.5f;
        v[i].y = rand() / (float)RAND_MAX - 0.5f;
        v[i].z = rand() / (float)RAND_MAX - 0.5f;
    }
}

int main()
{
    double tgpu = 0, tmem = 0;
    
    size_t size = sizeof(coord) * N;
    coord *p = (coord *)malloc(size);
    coord *v = (coord *)malloc(size);
    coord *d_p[2] = {NULL, NULL};
    coord *d_v[2] = {NULL, NULL};
    
    init_rand(p, N);
    init_rand(v, N);
    
    tmem = -wtime();
    hipMalloc((void **)&d_p[0], size);
    hipMalloc((void **)&d_p[1], size);
    hipMalloc((void **)&d_v[0], size);
    hipMalloc((void **)&d_v[1], size);
    hipMemcpy(d_p[0], p, size, hipMemcpyHostToDevice);
    hipMemcpy(d_v[0], v, size, hipMemcpyHostToDevice);
    tmem += wtime();
        
    tgpu = -wtime();
    int threadsPerBlock = 1024;
    dim3 block(threadsPerBlock);
    dim3 grid((N + threadsPerBlock - 1) / threadsPerBlock);    
    int index = 0;
    for (int i = 0; i < 2; i++, index ^= 1) {
        integrate<<<grid, block>>>(d_p[index ^ 1], d_v[index ^ 1], d_p[index], d_v[index], N, dt);
    }
    hipDeviceSynchronize();
    tgpu += wtime();
    
    tmem -= wtime();
    hipMemcpy(p, d_p[index], size, hipMemcpyDeviceToHost);
    hipMemcpy(v, d_v[index], size, hipMemcpyDeviceToHost);
    tmem += wtime();
    
    /*
    for (int i = 0; i < N; i++) {
        printf("%4d: %f %f %f %f %f %f\n", i, p[i].x, p[i].y, p[i].z, v[i].x, v[i].y, v[i].z);
    }
    */
    
    printf("sizeof(coord) = %d\n", sizeof(coord));
    printf("GPU version (sec.): %.6f\n", tgpu);
    printf("Memory ops. (sec.): %.6f\n", tmem);
    printf(" Total time (sec.): %.6f\n", tgpu + tmem);

    hipFree(d_p[0]);
    hipFree(d_p[1]);
    hipFree(d_v[0]);
    hipFree(d_v[1]);
    free(p);
    free(v);
    hipDeviceReset();
    return 0;
}

